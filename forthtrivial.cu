#include "hip/hip_runtime.h"
// Copyright 2024 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cstdint>
#define FORTH_CUSTOM_OPS

#include <array>

#include "forth.inc.h"
__device__ __host__ ForthOp Forth::GetOpKind(uint8_t c) {
  switch (c) {
    case 0x0:
      return ForthOp::kRead0;
    case 0x1:
      return ForthOp::kRead1;
    case 0x2:
      return ForthOp::kWrite0;
    case 0x3:
      return ForthOp::kWrite1;
    case 0x4:
      return ForthOp::kDup;
    case 0x5:
      return ForthOp::kDrop;
    case 0x6:
      return ForthOp::kSwap;
    case 0x7:
      return ForthOp::kIf0;
    case 0x8:
      return ForthOp::kInc;
    case 0x9:
      return ForthOp::kDec;
    case 0xA:
      return ForthOp::kAdd;
    case 0xB:
      return ForthOp::kSub;
    case 0xC:
      return ForthOp::kCopy0;
    case 0xD:
      return ForthOp::kCopy1;
    default:
      return (c >= 128 ? ForthOp::kJmp
                       : (c >= 64 ? ForthOp::kConst : ForthOp::kNoop));
  }
}

namespace {
const char *Forth::name() { return "forthtrivial"; }

void Forth::InitByteColors(
    std::array<std::array<uint8_t, 3>, 256> &byte_colors) {
  auto scale_color = [](std::array<uint8_t, 3> &color, size_t offset,
                        size_t num) {
    float darken_amount = offset * 0.2f / (num - 1);
    float multiplier = 1.0f - darken_amount;
    for (size_t c = 0; c < 3; c++) {
      color[c] =
          std::round(std::min(255.0f, std::max(0.0f, multiplier * color[c])));
    }
  };

  // I/O
  for (auto i : {0x00, 0x01, 0x02, 0x03, 0x0C, 0x0D}) {
    byte_colors[i] = {0x73, 0x01, 0xce};
  }
  // Stack manipulation
  for (auto i : {0x04, 0x05, 0x06, 0x08, 0x09, 0x0A, 0x0B}) {
    byte_colors[i] = {0x4e, 0x10, 0x01};
  }
  // Conditional jump
  byte_colors[0x07] = {0x00, 0x00, 0x00};
  // Forward jump
  for (size_t i = 0b10'000000; i < 0b11'000000; i++) {
    byte_colors[i] = {0x94, 0xd9, 0xff};
    scale_color(byte_colors[i], i - 0b10'000000, 0b1'000000);
  }
  // Backward jump
  for (size_t i = 0b11'000000; i < 0b100'000000; i++) {
    byte_colors[i] = {0xff, 0x77, 0x7d};
    scale_color(byte_colors[i], i - 0b11'000000, 0b1'000000);
  }
  // Constant
  for (size_t i = 0b01'000000; i < 0b10'000000; i++) {
    byte_colors[i] = {0xff, 0xff, 0xff};
    scale_color(byte_colors[i], i - 0b1'000000, 0b1'000000);
  }
  // Comment
  for (size_t i = 0x0E; i < 0x40; i++) {
    byte_colors[i] = {0x02, 0x8a, 0x37};
    scale_color(byte_colors[i], i - 0x0E, 50);
  }
}

__device__ void Forth::EvaluateOne(uint8_t *tape, int &pos, size_t &nops,
                                   Stack &stack) {
  // 000000xy (00-03) -> (read|write)(0|1)
  // 00000100 (04)    -> dup
  // 00000101 (05)    -> drop
  // 00000110 (06)    -> swap
  // 00000111 (07)    -> if0
  // 00001000 (08)    -> inc
  // 00001001 (09)    -> dec
  // 00001010 (0A)    -> add
  // 00001011 (0B)    -> sub
  // 0000110x (0C-0D) -> copy(0->1)(1->0)
  // 01xxxxxx (40-7F) -> stack.Push unsigned constant xxxxxx
  // 1Xxxxxxx (80-FF) -> jump to offset {+-}(xxxxxx+1)
  uint8_t command = tape[pos];
  switch (GetOpKind(command)) {
    case kRead0:
    case kRead1: {
      int t = command & 1;
      int addr = stack.Pop() % kSingleTapeSize;
      stack.Push(tape[(t ? kSingleTapeSize : 0) + addr]);
      break;
    }
    case kWrite0:
    case kWrite1: {
      int t = command & 1;
      int val = stack.Pop();
      int addr = stack.Pop() % kSingleTapeSize;
      tape[(t ? kSingleTapeSize : 0) + addr] = val;
      break;
    }
    case kDup: {
      int v = stack.Pop();
      stack.Push(v);
      stack.Push(v);
      break;
    }
    case kDrop:
      stack.Pop();
      break;
    case kSwap: {
      int a = stack.Pop();
      int b = stack.Pop();
      stack.Push(a);
      stack.Push(b);
      break;
    }
    case kIf0: {
      int v = stack.Pop();
      if (v) {
        pos++;
      }
      stack.Push(v);
      break;
    }
    case kInc: {
      stack.Push(stack.Pop() + 1);
      break;
    }
    case kDec: {
      stack.Push(stack.Pop() - 1);
      break;
    }
    case kAdd: {
      int a = stack.Pop();
      int b = stack.Pop();
      stack.Push(a + b);
      break;
    }
    case kSub: {
      int a = stack.Pop();
      int b = stack.Pop();
      stack.Push(a - b);
      break;
    }
    case kCopy0:
    case kCopy1: {
      int dir = command & 1;
      int addr = stack.Pop() % kSingleTapeSize;
      tape[(dir ? 0 : kSingleTapeSize) + addr] =
          tape[(dir ? kSingleTapeSize : 0) + addr];
      break;
    }
    case kConst: {
      stack.Push(command & 63);
      pos++;
      break;
    }
    case kJmp: {
      int abs = (command & 63) + 1;
      int jmp = command & 64 ? -abs : abs;
      pos += jmp;
      pos--;
      break;
    }
    default: {
      nops++;
    }
  }
  pos++;
}
}  // namespace
